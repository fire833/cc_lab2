

#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

const int arg_count = 12;
static const int mask_host[12] = { 11, 10, 9, 8, 7, 6, 5, 4, 3, 2, 1, 0 }; 

int *parse_input(char* input, int parsed_len) {
	int index = 0;
	int *output = (int *)calloc(parsed_len, sizeof(int));
	int sum = 0;
  	for (int i = 0; i < strlen(input); i++) {
    	// Shift over the sum and add a new value, whatever it may be.
    	switch (input[i]) {
    	case '0':
      		sum = 10 * sum;
      		continue;
    	case '1':
      		sum = 10 * sum + 1;
      		continue;
    	case '2':
      		sum = 10 * sum + 2;
      		continue;
    	case '3':
      		sum = 10 * sum + 3;
      		continue;
    	case '4':
      		sum = 10 * sum + 4;
      		continue;
    	case '5':
      		sum = 10 * sum + 5;
      		continue;
    	case '6':
      		sum = 10 * sum + 6;
      		continue;
    	case '7':
      		sum = 10 * sum + 7;
      		continue;
    	case '8':
      		sum = 10 * sum + 8;
      		continue;
    	case '9':
      		sum = 10 * sum + 9;
      		continue;

    	// We are at the end of the number, reset to a new sum.
    	case ',': {
      		output[index] = sum;
      		sum = 0;
      		index++;
     		continue;
    	}
    	}
  	}

  	output[index] = sum;
	return output;
}

// Kernel to permute values
__global__ void permute_array(int *in, int* mask, int *out) {
    unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < arg_count) {
        out[mask[index]] = in[index];
    }
}

__host__ int main(int argc, char **argv) {
    if (argc != 3) {
		printf("{\"error\": \"2 arguments required, the program call name, the number of values (as an integer), and the list of values, comma separated.\",\"code\":1}");
		exit(1);
	}

	char *none;
  	int input_int_len = strtol(argv[1], &none, 10);

	if (input_int_len != arg_count) {
		printf("{\"error\": \"must provide as many inputs as there are arguments (%d)\",\"code\":1}", arg_count);
		exit(1);
	}

    int *input_host = parse_input(argv[2], input_int_len);
    int *output_host = (int*)calloc(input_int_len, sizeof(int));
    int *input_gpu;
    int *mask_gpu;
    int *output_gpu;

    hipMalloc((void**) &input_gpu, input_int_len * sizeof(int));
    hipMalloc((void**) &mask_gpu, input_int_len * sizeof(int));
    hipMalloc((void**) &output_gpu, input_int_len * sizeof(int));

    // Copy data to device
    hipMemcpy(input_gpu, input_host, input_int_len * sizeof(int), hipMemcpyHostToDevice);
    // Copy mask to device
    hipMemcpy(mask_gpu, mask_host, input_int_len * sizeof(int), hipMemcpyHostToDevice);
    
    clock_t start, end;
    
	start = clock();
    permute_array<<<1, input_int_len>>>(input_gpu, mask_gpu, output_gpu);
    hipDeviceSynchronize();
	end = clock();

    // Copy data back over
    hipMemcpy(output_host, output_gpu, input_int_len * sizeof(int), hipMemcpyDeviceToHost);

	printf("{\"values\": [");
	for (int i = 0; i < input_int_len; i++) {
		if (i == input_int_len - 1) {
			printf("%d],", output_host[i]);
		} else {
    		printf("%d,", output_host[i]);
		}
  	}

	printf("\"compute\": %.1f, \"code\": 0}\\n", ((double) (end - start)));

    hipFree(&input_gpu);
    hipFree(&output_gpu);
    hipFree(&mask_gpu);
    free(input_host);
    free(output_host);
}
